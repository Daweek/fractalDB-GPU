#include "hip/hip_runtime.h"
/*
 * Accel.cu
 *
 *  Created on: May 31, 2021
 *      Author: Edg@r j.
 */
#include "Accel.cuh"

__global__ void find_borders_kernel(float2* array, float4 *brd, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cacheMaxX[256];
	__shared__ float cacheMaxY[256];
	__shared__ float cacheMinX[256];
	__shared__ float cacheMinY[256];


	float maxX = 1.0;
	float maxY = 1.0;
	float minX = -1.0;
	float minY = -1.0;


	while(index + offset < n){
		maxX = fmaxf(maxX, array[index + offset].x);
		maxY = fmaxf(maxY, array[index + offset].y);

		minX = fminf(minX, array[index + offset].x);
		minY = fminf(minY, array[index + offset].y);

		offset += stride;
	}

	cacheMaxX[threadIdx.x] = maxX;
	cacheMaxY[threadIdx.x] = maxY;
	cacheMinX[threadIdx.x] = minX;
	cacheMinY[threadIdx.x] = minY;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cacheMaxX[threadIdx.x] = fmaxf(cacheMaxX[threadIdx.x], cacheMaxX[threadIdx.x + i]);
			cacheMaxY[threadIdx.x] = fmaxf(cacheMaxY[threadIdx.x], cacheMaxY[threadIdx.x + i]);
			cacheMinX[threadIdx.x] = fminf(cacheMinX[threadIdx.x], cacheMinX[threadIdx.x + i]);
			cacheMinY[threadIdx.x] = fminf(cacheMinY[threadIdx.x], cacheMinY[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		brd[0].x = fmaxf(brd[0].x, cacheMaxX[0]);
		brd[0].y = fmaxf(brd[0].y, cacheMaxY[0]);
		brd[0].z = fminf(brd[0].z, cacheMinX[0]);
		brd[0].w = fminf(brd[0].w, cacheMinY[0]);
		atomicExch(mutex, 0);  //unlock
	}
}

__global__ void kernel_2(float2* d_poss,float2* d_color , int numPoints,mapping *d_mappings, int numMappings)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // If needed for performance, move hiprand_init to seperate kernel and store
  // states in device memory
  hiprandState state;
  hiprand_init((unsigned long long) clock(), index, 0, &state);

  // Set up transformation mapping once per block in shared memory
  extern __shared__ mapping maps[];
  if(threadIdx.x == 0)
  {
    #pragma unroll
    for(int i = 0; i < numMappings; i++)
        maps[i] = d_mappings[i];
  }
  __syncthreads();

  // Initially start at a mapping vertex to guarantee we stay inside the
  // iterated function system
  int currentTarget = index % numMappings;
  float2 currentPosition, newPosition;
  currentPosition.x = maps[currentTarget].x;
  currentPosition.y = maps[currentTarget].y;

  for(int i = index; i < numPoints; i += stride)
  {
    // set the current vertex to the currentPosition
    d_poss[i].x = currentPosition.x ;
    d_poss[i].y = currentPosition.y ;

    // set the iteration percentage and current target mapping
    d_color[i].x =  i / (float) numPoints;
    d_color[i].y = currentTarget;

    // find random target with given mapping probabilities
    // If needed for performance, find method to remove thread divergence
    // Note: changing 4 to numMappings in for loop reduced performance 50%
    float currentProb = hiprand_uniform(&state);
    float totalProb = 0.0f;
    for(int j = 0; j < numMappings; j++)
    {
        totalProb += maps[j].p;
        if(currentProb < totalProb)
        {
            currentTarget = j;
            break;
        }
    }

    // calculate the transformation
    // (x_n+1) = (a b)(x_n) + (e)
    // (y_n+1)   (c d)(y_n)   (f)
    newPosition.x = maps[currentTarget].a * currentPosition.x +
                    maps[currentTarget].b * currentPosition.y +
                    maps[currentTarget].x;
    newPosition.y = maps[currentTarget].c * currentPosition.x +
                    maps[currentTarget].d * currentPosition.y +
                    maps[currentTarget].y;
    currentPosition = newPosition;
  }

}

__global__ void kernel_test(float2* d_pointData, int numPoints,mapping *d_mappings, int numMappings)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //int stride = blockDim.x * gridDim.x;

  int currentTarget = index % numMappings;

  //d_pointData[index].x = 0.0f + currentTarget * 0.5f;
  d_pointData[index].y = 0.0f + currentTarget * 0.10f;
  //d_pointData[index].x = 0.0f;
  //d_pointData[index].y = 0.0f;

}


Accel::Accel() {

	// Initialize CUDA
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipGetDevice(&m_cuDevice));
	checkCudaErrors(hipGetDeviceProperties(&m_cuDevProp,m_cuDevice));

	hipDriverGetVersion(&m_driverVersion);
	hipRuntimeGetVersion(&m_runtimeVersion);

	// Print device properties
	printf("\tDevice Name: %s\n", m_cuDevProp.name);
	printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
					m_driverVersion / 1000, (m_driverVersion % 100) / 10,
					m_runtimeVersion / 1000, (m_runtimeVersion % 100) / 10);
	printf("\tCompute Capability: %d.%d\n", m_cuDevProp.major, m_cuDevProp.minor);
	printf("\tTotal Global Memory: %ld bytes\n", m_cuDevProp.totalGlobalMem);
	printf("\tNumber of Multiprocessors: %d\n", m_cuDevProp.multiProcessorCount);
	printf("\tMaximum Threads per Multiprocessor: %d\n",
		m_cuDevProp.maxThreadsPerMultiProcessor);
	printf("\tTotal Number of Threads: %d\n", m_cuDevProp.multiProcessorCount *
		m_cuDevProp.maxThreadsPerMultiProcessor);
	printf("\tMaximum Threads per Block: %d\n", m_cuDevProp.maxThreadsPerBlock);
	

	// Setting up all pointers
	d_glmap = d_glPoss = d_glColor = NULL;
	d_map		= NULL;
	g_strucMapVBO		= NULL;
	d_borders = NULL;
	h_borders = NULL;
	d_mutex = NULL;
	
	// CUDA related structs
	g_mapVBO = 0;

	// Timer related
	m_fFlops = m_fStepsec = 0.0f;

	// Memory Flags related
	m_bChangeInterop = m_bChangeMalloc = true;

	m_numBlocks = m_blockSize = 0;
	m_fXmax = m_fXmin = m_fYmax = m_fYmin = 0.0;
}

void Accel::interopCUDA(){
	std::cout<<"Seting up CUDA-OpenGL buffer...\n";
  // Prepare graphics interoperability
  if(g_strucMapVBO != NULL) 
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucMapVBO));

	if(g_strucPoss != NULL) 
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucPoss));

	if(g_strucColor != NULL) 
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucColor));


  glDeleteBuffers(1,&g_mapVBO);
	glDeleteBuffers(1,&g_poss);
	glDeleteBuffers(1,&g_color);

  // Creation of share buffer between CUDA and OpenGL
  // For mapping position and color
  glGenBuffers(1, &g_mapVBO);
  glBindBuffer(GL_ARRAY_BUFFER, g_mapVBO);
  unsigned int size = MAX_POINTS * 4 * sizeof(float);
  glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

	glGenBuffers(1, &g_poss);
  glBindBuffer(GL_ARRAY_BUFFER, g_poss);
  unsigned int sizeP = MAX_POINTS * 2 * sizeof(float);
  glBufferData(GL_ARRAY_BUFFER, sizeP, 0, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

	glGenBuffers(1, &g_color);
  glBindBuffer(GL_ARRAY_BUFFER, g_color);
  unsigned int sizeC = MAX_POINTS * 2 * sizeof(float);
  glBufferData(GL_ARRAY_BUFFER, sizeC, 0, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

  // Register CUDA and OpenGL Interop
  checkCudaErrors(hipGraphicsGLRegisterBuffer(&g_strucMapVBO,g_mapVBO,cudaGraphicsMapFlagsNone));
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&g_strucPoss,g_poss,cudaGraphicsMapFlagsNone));
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&g_strucColor,g_color,cudaGraphicsMapFlagsNone));

}


void Accel::malloCUDA(mapping *mapped, int numMaps){

	// For the image buffer
	if(d_map != NULL)
		checkCudaErrors(hipFree(d_map));
  checkCudaErrors(hipMalloc((void**)&d_map,numMaps*sizeof(mapping)));
  checkCudaErrors(hipMemcpy(d_map,mapped,numMaps*sizeof(mapping),hipMemcpyHostToDevice));

	// To check borders
	if(d_borders != NULL)
		checkCudaErrors(hipFree(d_borders));
	hipMalloc((void**)&d_borders,sizeof(float4));
	hipMemset(d_borders,0, sizeof(float4));

	if(d_mutex != NULL)
		checkCudaErrors(hipFree(d_mutex));
	hipMalloc((void**)&d_mutex,sizeof(int));
	hipMemset(d_mutex, 0, sizeof(int));


	if(h_borders != NULL)
		free(h_borders);
	h_borders = (float*)malloc(4*sizeof(float)); 

}

void Accel::fractalKernel(int numMappings, int numPoints){

	m_numBlocks = 1;
	m_blockSize	= 1024;

  size_t mapsizevbo;
  checkCudaErrors(hipGraphicsMapResources(1,&g_strucMapVBO,0));
  checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_glmap,&mapsizevbo,g_strucMapVBO));

  checkCudaErrors(hipGraphicsMapResources(1,&g_strucPoss,0));
  checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_glPoss,&mapsizevbo,g_strucPoss));

	checkCudaErrors(hipGraphicsMapResources(1,&g_strucColor,0));
  checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_glColor,&mapsizevbo,g_strucColor));
  
  hipEvent_t start, stop;
  checkCudaErrors( hipEventCreate(&start) );
  checkCudaErrors( hipEventCreate(&stop) );
  
  checkCudaErrors( hipEventRecord(start) );
  
	
	// Compute Fractal
		kernel_2<<<m_numBlocks, m_blockSize, numMappings * sizeof(mapping)>>>
      ((float2*)d_glPoss,(float2*)d_glColor , numPoints, d_map, numMappings);	
	
	// Compute Borders of the fractal
	
	dim3 gridSize = 256;
	dim3 blockSize = 256;

	hipMemset(d_mutex, 0, sizeof(int));

  
		find_borders_kernel<<< gridSize, blockSize >>>
			((float2*)d_glPoss,d_borders, d_mutex, (unsigned int)numPoints);
		   
	checkCudaErrors(hipMemcpy(h_borders, d_borders, sizeof(float4), hipMemcpyDeviceToHost));

	m_fXmax = h_borders[0];
	m_fYmax = h_borders[1];
	m_fXmin = h_borders[2];
	m_fYmin = h_borders[3];
	
	/*
		cout<<"Maximum X found on gpu was: "<<m_fXmax<<endl;
		cout<<"Maximum Y found on gpu was: "<<m_fYmax<<endl;
		cout<<"Minimum X found on gpu was: "<<m_fXmin<<endl;
		cout<<"Minimum Y found on gpu was: "<<m_fYmin<<endl<<endl;
	*/	

  checkCudaErrors( hipEventRecord(stop) );

  // handle any synchronous and asynchronous kernel errors
  checkCudaErrors( hipGetLastError() );
  checkCudaErrors( hipDeviceSynchronize() );

  // record and print kernel timing
  checkCudaErrors( hipEventSynchronize(stop) );
  m_kernel_mili = 0;
  checkCudaErrors( hipEventElapsedTime(&m_kernel_mili, start, stop) );

  // Unmap OpenGL resources
  checkCudaErrors(hipGraphicsUnmapResources(1,&g_strucMapVBO,0));
	checkCudaErrors(hipGraphicsUnmapResources(1,&g_strucPoss,0));
	checkCudaErrors(hipGraphicsUnmapResources(1,&g_strucColor,0));
	
}

Accel::~Accel() {
	// Unregister if CUDA-InteropGL
	std::cout<<"Unregistering CUDA-GL Resources...\n";

	if(g_strucMapVBO  != NULL)
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucMapVBO));

	if(g_strucPoss != NULL) 
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucPoss));

	if(g_strucColor != NULL) 
		checkCudaErrors(hipGraphicsUnregisterResource(g_strucColor));
	

	// Free memory for HALF interop
	//delete [] m_fPossVBO;
}

